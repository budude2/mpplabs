#include <opencv2/opencv.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgcodecs.hpp>
#include <iostream>
#include "kernel.cu"
#include "support.h"

int main()
{
    std::vector<cv::Mat> img;
    Timer timer;

    std::cout << "Opening images....";
    startTime(&timer);

    img.push_back(cv::imread("../noise_set2/noise1.jpg", 1));
    img.push_back(cv::imread("../noise_set2/noise2.jpg", 1));
    img.push_back(cv::imread("../noise_set2/noise3.jpg", 1));
    img.push_back(cv::imread("../noise_set2/noise4.jpg", 1));
    img.push_back(cv::imread("../noise_set2/noise5.jpg", 1));

    cv::Mat res(img[0].rows, img[0].cols, CV_8UC3);

    unsigned int vecSize = img.size();
    unsigned int imageSize = img[0].step * img[0].rows;
    unsigned int blockSize = imageSize * vecSize;

    unsigned char **images = new unsigned char*[vecSize];
    unsigned char *imageData = new unsigned char[blockSize];

    for(unsigned int i = 0; i < vecSize; i++)
    {
        images[i] = img[i].data;
    }

    for(unsigned int j = 0; j < vecSize; j++)
    {
        for(unsigned int i = 0; i < imageSize; i++)
        {
            imageData[j * imageSize + i] = images[j][i];
        }
    }

    stopTime(&timer);
    std::cout << elapsedTime(timer) << " s" << std::endl;

    std::cout << "Allocate arrays...";
    startTime(&timer);

    const int resSize   = res.step * res.rows;

    unsigned char *d_res;
    unsigned char *img_d;

    hipMalloc<unsigned char>(&img_d, blockSize);
    hipMalloc<unsigned char>(&d_res, resSize);

    stopTime(&timer);
    std::cout << elapsedTime(timer) << " s" << std::endl;

    std::cout << "Copy images.......";
    startTime(&timer);

    hipMemcpy(img_d, imageData, blockSize, hipMemcpyHostToDevice);

    stopTime(&timer);
    std::cout << elapsedTime(timer) << " s" << std::endl;

    std::cout << "Launch kernel.....";
    startTime(&timer);

    const dim3 block(16,16);
    const dim3 grid((img[0].cols + block.x - 1)/block.x, (img[0].rows + block.y - 1)/block.y);

    image_proc<<<grid, block>>>(img_d, d_res, img[0].cols, img[0].rows, img[0].step, img[0].step * img[0].rows, img.size());

    stopTime(&timer);
    std::cout << elapsedTime(timer) << " s" << std::endl;

    std::cout << "Copy result.......";
    startTime(&timer);

    hipMemcpy(res.ptr(), d_res, resSize, hipMemcpyDeviceToHost);

    stopTime(&timer);
    std::cout << elapsedTime(timer) << " s" << std::endl;

    std::cout << "Write result......";
    startTime(&timer);

    std::vector<int> compression_param;
    compression_param.push_back(cv::IMWRITE_JPEG_QUALITY);
    compression_param.push_back(100);

    cv::imwrite("result.jpg", res, compression_param);

    stopTime(&timer);
    std::cout << elapsedTime(timer) << " s" << std::endl;

    delete[] images;
    delete[] imageData;
    res.release();
    hipFree(img_d);
    hipFree(d_res);
}
