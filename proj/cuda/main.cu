#include <opencv2/opencv.hpp>
#include <iostream>
#include "kernel.cu"
#include "support.h"

int main(int argc, char* argv[])
{
    if (argc < 2)
    {
        std::cout << "Usage: avg [path to images]\n";
        return 1;
    }

    std::vector<cv::Mat> img;
    Timer timer;

    startTime(&timer);

    img = loadFiles(argv);

    cv::Mat res(img[0].rows, img[0].cols, CV_8UC3);

    unsigned int vecSize = img.size();
    unsigned int imageSize = img[0].step * img[0].rows;
    unsigned int blockSize = imageSize * vecSize;

    unsigned char **images = new unsigned char*[vecSize];
    unsigned char *imageData = new unsigned char[blockSize];

    std::cout << "\nOpening images....";
    stopTime(&timer);
    std::cout << elapsedTime(timer) << " s" << std::endl;

    startTime(&timer);
    for(unsigned int i = 0; i < vecSize; i++)
    {
        images[i] = img[i].data;
    }

    for(unsigned int j = 0; j < vecSize; j++)
    {
        for(unsigned int i = 0; i < imageSize; i++)
        {
            imageData[j * imageSize + i] = images[j][i];
        }
    }

    std::cout << "Images -> block...";
    stopTime(&timer);
    std::cout << elapsedTime(timer) << " s" << std::endl;

    std::cout << "Allocate arrays...";
    startTime(&timer);

    const int resSize   = res.step * res.rows;

    unsigned char *d_res;
    unsigned char *img_d;

    hipMalloc<unsigned char>(&img_d, blockSize);
    hipMalloc<unsigned char>(&d_res, resSize);

    stopTime(&timer);
    std::cout << elapsedTime(timer) << " s" << std::endl;

    std::cout << "Copy images.......";
    startTime(&timer);

    hipMemcpy(img_d, imageData, blockSize, hipMemcpyHostToDevice);

    stopTime(&timer);
    std::cout << elapsedTime(timer) << " s" << std::endl;

    std::cout << "Launch kernel.....";
    startTime(&timer);

    const dim3 block(16,16);
    const dim3 grid((img[0].cols + block.x - 1)/block.x, (img[0].rows + block.y - 1)/block.y);

    image_proc<<<grid, block>>>(img_d, d_res, img[0].cols, img[0].rows, img[0].step, img[0].step * img[0].rows, img.size());

    stopTime(&timer);
    std::cout << elapsedTime(timer) << " s" << std::endl;

    std::cout << "Copy result.......";
    startTime(&timer);

    hipMemcpy(res.ptr(), d_res, resSize, hipMemcpyDeviceToHost);

    stopTime(&timer);
    std::cout << elapsedTime(timer) << " s" << std::endl;

    std::cout << "Write result......";
    startTime(&timer);

    std::vector<int> compression_param;
    compression_param.push_back(cv::IMWRITE_JPEG_QUALITY);
    compression_param.push_back(100);

    cv::imwrite("result.jpg", res, compression_param);

    stopTime(&timer);
    std::cout << elapsedTime(timer) << " s" << std::endl;

    delete[] images;
    delete[] imageData;
    res.release();
    hipFree(img_d);
    hipFree(d_res);
}
